// Simple vector addition, from the samples provided in the CUDA SDK.
// Author: Allen Porter <allen@thebends.org>

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void VecAdd(float* A, float* B, float* C) {
  int i = threadIdx.x;
  C[i] = A[i] + B[i];
}

int main(int argc, char**argv)
{
  int N = 10;
  size_t size = N * sizeof(float);

  // Input; Host memory
  float* h_A = (float*)malloc(size);
  float* h_B = (float*)malloc(size);
  for (int i = 0; i < N; i++) {
    h_A[i] = i;
    h_B[i] = i;
  }


  // Device memory
  float* d_A;
  hipMalloc((void**)&d_A, size);
  float* d_B;
  hipMalloc((void**)&d_B, size);
  float* d_C;
  hipMalloc((void**)&d_C, size);

  // Copy from host to device
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // Invoke kernel
  VecAdd<<<1, N>>>(d_A, d_B, d_C);

  float* h_C = (float*)malloc(size);
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++) {
    printf("%0.f ", h_A[i]);
  }
  printf("\n");
  for (int i = 0; i < N; i++) {
    printf("%0.f ", h_B[i]);
  }
  printf("\n");
  for (int i = 0; i < N; i++) {
    printf("%0.f ", h_C[i]);
  }
  printf("\n");

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  return 0;
}
