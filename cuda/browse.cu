#include "hip/hip_runtime.h"
// Tries to dump memory in some far off location just to see whats there.
// Author: Allen Porter <allen@thebends.org>

#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdlib.h>
#include <stdio.h>

static const char* out_filename = "OUT";

__global__ void Browse(unsigned long* base, unsigned long* out) {
  int i = threadIdx.x;
  unsigned long* b = (unsigned long*)0x20000000LL;
  out[i] = [i];
}

int main(int argc, char**argv)
{
  // Read 4 mb of data
  int N = 1024 * 1024;
  size_t size = N * sizeof(long);

  // A base address
  unsigned long* d_base;
  hipMalloc((void**)&d_base, 1);

  unsigned long* d_out;
  hipMalloc((void**)&d_out, size);

  printf("Invoking kernel\n");

  // Invoke kernel
  Browse<<<1, N>>>(d_base, d_out);

  hipDeviceSynchronize();

  unsigned long* h_out = (unsigned long*)malloc(size);
  printf("Default values\n");
  memset(h_out, 'A', size);

  printf("Copying results\n");
  hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

  printf("Writing to ./%s\n", out_filename);
  FILE* f = fopen(out_filename, "w");
  if (!f) {
    perror("fopen");
    return 1;
  }
  size_t result = fwrite(h_out, N, sizeof(long), f);
  if (result <= 0) {
    perror("fwrite");
    return 1;
  }
  fclose(f);
  return 0;
}
